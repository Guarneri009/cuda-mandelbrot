#include "hip/hip_runtime.h"
#include <iostream>
#include <cstddef>
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdint>

constexpr auto threads_perblock = 1024;

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        std::cout << hipGetErrorString(err) << "  " << file << "  " << line << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

inline __device__ float fminf(float a, float b)
{
    return a < b ? a : b;
}

inline __device__ float fmaxf(float a, float b)
{
    return a > b ? a : b;
}

inline __device__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

inline __device__ void hsv2rgb_gpu(int h, int s, int v, uint8_t *r, uint8_t *g, uint8_t *b)
{
    if (h < 0)
        h += (1 - h / 360) * 360;
    if (360 < h)
        h %= 360;
    auto h1 = (h * 4096 + 50) / 120;
    auto s1 = (s * 4096 + 50) / 100;
    auto v1 = (v * 4096 + 50) / 100;
    auto h2 = h1 % 4096;
    auto a1{0}, a2{0};
    if (h2 < 2048)
    {
        a1 = (4096 - (2048 - h2) * s1 / 2048) * v1 / 4096;
        a2 = v1;
    }
    else
    {
        a2 = (4096 - (h2 - 2048) * s1 / 2048) * v1 / 4096;
        a1 = v1;
    }

    auto b1 = clamp((a2 * 255 + 2048) / 4096, 0, 255);
    auto b2 = clamp((a1 * 255 + 2048) / 4096, 0, 255);
    auto b3 = clamp(((4096 - s1) * v1 / 4096 * 255 + 2048) / 4096, 0, 255);

    switch (h1 / 4096)
    {
    case 1:
        *g = b1;
        *b = b2;
        *r = b3;
        break;
    case 2:
        *b = b1;
        *r = b2;
        *g = b3;
        break;
    default:
        *r = b1;
        *g = b2;
        *b = b3;
        break;
    }
}

inline __device__ int recurrence_formula(thrust::complex<float> c, thrust::complex<float> z, int max_iter)
{
    // mandelbrot recurrence formula
    for (int i = 0; i < max_iter; i++)
    {
        z = z * z + c;
        if (thrust::norm(z) > 2.0f)
            return i % 360;
    }
    return 0;
}

inline __device__ int mandelbrot(int x, int y, int view_size, float scale, float center_x, float center_y)
{
    auto center = static_cast<float>(view_size) / 2.0f;
    auto jx = (static_cast<float>(x) - center) / center * scale - center_x;
    auto jy = (center - static_cast<float>(y)) / center * scale + center_y;
    thrust::complex<float> c(jx, jy);
    thrust::complex<float> z(0, 0);
    return recurrence_formula(c, z, 1000);
}

__global__ void kernel(unsigned char *ptr, int view_size, float scale, float center_x, float center_y)
{
    uint8_t r{0}, g{0}, b{0};
    auto x = blockIdx.x;
    auto y = threadIdx.x;
    auto offset = x + y * gridDim.x;
    auto value = mandelbrot(x, y, view_size, scale, center_x, center_y);
    hsv2rgb_gpu(value, 100, 100, &r, &g, &b);
    uint8_t rgba[4] = {r, g, b, 255};
    memcpy(ptr + offset * 4, rgba, 4);
}

extern "C" unsigned char *MandelbrotGPU(std::size_t size, int view_size, float scale, float center_x, float center_y)
{
    unsigned char *ptr_gpu;

    HANDLE_ERROR(hipMalloc((void **)&ptr_gpu, size));

    auto *ptr = new unsigned char[size];
    HANDLE_ERROR(hipMemcpy(ptr_gpu, ptr, size, hipMemcpyHostToDevice));

    auto blocks_per_grid = ((view_size * view_size) + threads_perblock - 1) / threads_perblock;
    std::cout << "CUDA kernel [" << blocks_per_grid << "] blocks [" << threads_perblock << "] threads" << std::endl;
    kernel<<<blocks_per_grid, threads_perblock>>>(ptr_gpu, view_size, scale, center_x, center_y);

    HANDLE_ERROR(hipMemcpy(ptr, ptr_gpu, size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(ptr_gpu));
    return ptr;
}
